
#include <hip/hip_runtime.h>
#include <stdio.h>
#define REAL float

__global__ void initFun(int* nf, int devnum){
  int n=threadIdx.x + blockIdx.x*blockDim.x;//+
        //devnum*blockDim.x*gridDim.x;
  nf[n]*=10;
  //printf("%d\t%d\t%d\t%d\t%d\n", n, nf[n],devnum, gridDim.x,blockDim.x);
}

int main(int argc, char* argv[]){
  if(argc<2){
  fprintf(stderr, "USAGE: main <num_of_devices>" 
                   "<device_indices>\n");
  return -1;
  }
  
  int* info_devs=(int*)calloc(argc-1, sizeof(int));
  
  info_devs[0]=atoi(argv[1]);
  for(int i=1;i<argc-1;i++){
    info_devs[i]=atoi(argv[i+1]);
  }
  
  fprintf(stderr,"num of devices: %d\n",info_devs[0]);
  for(int i=1;i<argc-1;i++)
          fprintf(stderr,"i_d=%d\n",info_devs[i]);
  
  int N=3*(1<<6);

  hipStream_t* streams;
  int** nfd=(int**)calloc(info_devs[0], sizeof(int*));
  int** nfh=(int**)calloc(info_devs[0], sizeof(int*));
  //int* Nfh;
  //cudaMallocHost((void**)&Nfh, N*sizeof(int));
  
  streams=(hipStream_t*)calloc(info_devs[0], sizeof(hipStream_t));
  
  for(int i=0;i<info_devs[0];i++){
     hipSetDevice(info_devs[i+1]);
     hipStreamCreate(&streams[i]);

     hipMalloc((void**)&nfd[i], (N/info_devs[0])*sizeof(int));
     hipHostMalloc((void**)&nfh[i], (N/info_devs[0])*sizeof(int), hipHostMallocDefault);
     
     for(int n=0;n<N/info_devs[0]; n++)
        nfh[i][n]=n+i*N/info_devs[0];

     hipMemcpyAsync(nfd[i],nfh[i], 
                    (N/info_devs[0])*sizeof(int),
                    hipMemcpyHostToDevice,streams[i]);
       
        
     initFun<<<N/info_devs[0]/32, 32, 0, streams[i]>>>(nfd[i],i);
        
//      cudaMemcpyAsync(Nfh+i*N/info_devs[0],nfd[i], 
//                     (N/info_devs[0])*sizeof(int),
//                     cudaMemcpyDeviceToHost,streams[i]);

     hipMemcpyAsync(nfh[i],nfd[i], 
                    (N/info_devs[0])*sizeof(int),
                    hipMemcpyDeviceToHost,streams[i]);

  }       
  
  for(int i=0;i<info_devs[0];i++){
     hipSetDevice(info_devs[i+1]);
     hipStreamSynchronize(streams[i]);
     
     for(int n=0;n<N/info_devs[0];n++)
       fprintf(stderr,"nfh[%d][%d]=%d\n",i,n, nfh[i][n]);
  }
  
  for(int i=0;i<info_devs[0];i++){   
     hipStreamDestroy(streams[i]);
     hipFree(nfd[i]);
     hipHostFree(nfh[i]);
     hipDeviceReset();
   }
   //cudaFreeHost(Nfh);

   return 0;
} 
